#include "hip/hip_runtime.h"
/*
  Copyright 2022-2023 SINTEF AS

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <opm/common/ErrorMacros.hpp>
#include <opm/simulators/linalg/cuistl/detail/cusparse_matrix_operations.hpp>
#include <stdexcept>
namespace Opm::cuistl::detail
{
namespace
{

    // TODO: figure out if this can be generalized effectively, this seems excessively verbose
    // explicit formulas based on Dune cpu code
    template <class T, int blocksize>
    __device__ __forceinline__ void invBlockOutOfPlace(const T* __restrict__ srcBlock, T* __restrict__ dstBlock)
    {
        if (blocksize == 1) {
            dstBlock[0] = 1.0 / (srcBlock[0]);
        } else if (blocksize == 2) {
            T detInv = 1.0 / (srcBlock[0] * srcBlock[3] - srcBlock[1] * srcBlock[2]);
            dstBlock[0] = srcBlock[3] * detInv;
            dstBlock[1] = -srcBlock[1] * detInv;
            dstBlock[2] = -srcBlock[2] * detInv;
            dstBlock[3] = srcBlock[0] * detInv;
        } else if (blocksize == 3) {
            // based on Dune implementation
            T t4 = srcBlock[0] * srcBlock[4];
            T t6 = srcBlock[0] * srcBlock[5];
            T t8 = srcBlock[1] * srcBlock[3];
            T t10 = srcBlock[2] * srcBlock[3];
            T t12 = srcBlock[1] * srcBlock[6];
            T t14 = srcBlock[2] * srcBlock[6];

            T t17 = 1.0
                / (t4 * srcBlock[8] - t6 * srcBlock[7] - t8 * srcBlock[8] + t10 * srcBlock[7] + t12 * srcBlock[5]
                   - t14 * srcBlock[4]); // t17 is 1/determinant

            dstBlock[0] = (srcBlock[4] * srcBlock[8] - srcBlock[5] * srcBlock[7]) * t17;
            dstBlock[1] = -(srcBlock[1] * srcBlock[8] - srcBlock[2] * srcBlock[7]) * t17;
            dstBlock[2] = (srcBlock[1] * srcBlock[5] - srcBlock[2] * srcBlock[4]) * t17;
            dstBlock[3] = -(srcBlock[3] * srcBlock[8] - srcBlock[5] * srcBlock[6]) * t17;
            dstBlock[4] = (srcBlock[0] * srcBlock[8] - t14) * t17;
            dstBlock[5] = -(t6 - t10) * t17;
            dstBlock[6] = (srcBlock[3] * srcBlock[7] - srcBlock[4] * srcBlock[6]) * t17;
            dstBlock[7] = -(srcBlock[0] * srcBlock[7] - t12) * t17;
            dstBlock[8] = (t4 - t8) * t17;
        }
    }

    // explicit formulas based on Dune cpu code
    template <class T, int blocksize>
    __device__ __forceinline__ void invBlockInPlace(T* __restrict__ block)
    {
        if (blocksize == 1) {
            block[0] = 1.0 / (block[0]);
        } else if (blocksize == 2) {
            T detInv = 1.0 / (block[0] * block[3] - block[1] * block[2]);

            T temp = block[0];
            block[0] = block[3] * detInv;
            block[1] = -block[1] * detInv;
            block[2] = -block[2] * detInv;
            block[3] = temp * detInv;
        } else if (blocksize == 3) {
            T t4 = block[0] * block[4];
            T t6 = block[0] * block[5];
            T t8 = block[1] * block[3];
            T t10 = block[2] * block[3];
            T t12 = block[1] * block[6];
            T t14 = block[2] * block[6];

            T det = (t4 * block[8] - t6 * block[7] - t8 * block[8] + t10 * block[7] + t12 * block[5] - t14 * block[4]);
            T t17 = T(1.0) / det;

            T matrix01 = block[1];
            T matrix00 = block[0];
            T matrix10 = block[3];
            T matrix11 = block[4];

            block[0] = (block[4] * block[8] - block[5] * block[7]) * t17;
            block[1] = -(block[1] * block[8] - block[2] * block[7]) * t17;
            block[2] = (matrix01 * block[5] - block[2] * block[4]) * t17;
            block[3] = -(block[3] * block[8] - block[5] * block[6]) * t17;
            block[4] = (matrix00 * block[8] - t14) * t17;
            block[5] = -(t6 - t10) * t17;
            block[6] = (matrix10 * block[7] - matrix11 * block[6]) * t17;
            block[7] = -(matrix00 * block[7] - t12) * t17;
            block[8] = (t4 - t8) * t17;
        }
    }

    enum class MVType { SET, PLUS, MINUS };
    // SET:   c  = A*b
    // PLS:   c += A*b
    // MINUS: c -= A*b
    template <class T, int blocksize, MVType OP>
    __device__ __forceinline__ void matrixVectorProductWithAction(const T* A, const T* b, T* c)
    {
        for (int i = 0; i < blocksize; ++i) {
            if (OP == MVType::SET) {
                c[i] = 0;
            }

            for (int j = 0; j < blocksize; ++j) {
                if (OP == MVType::SET || OP == MVType::PLUS) {
                    c[i] += A[i * blocksize + j] * b[j];
                } else if (OP == MVType::MINUS) {
                    c[i] -= A[i * blocksize + j] * b[j];
                }
            }
        }
    }

    template <class T, int blocksize>
    __device__ __forceinline__ void mv(const T* a, const T* b, T* c)
    {
        matrixVectorProductWithAction<T, blocksize, MVType::SET>(a, b, c);
    }

    template <class T, int blocksize>
    __device__ __forceinline__ void umv(const T* a, const T* b, T* c)
    {
        matrixVectorProductWithAction<T, blocksize, MVType::PLUS>(a, b, c);
    }

    template <class T, int blocksize>
    __device__ __forceinline__ void mmv(const T* a, const T* b, T* c)
    {
        matrixVectorProductWithAction<T, blocksize, MVType::MINUS>(a, b, c);
    }

    // dst -= A*B*C
    template <class T, int blocksize>
    __device__ __forceinline__ void mmx2Subtraction(T* A, T* B, T* C, T* dst)
    {

        T tmp[blocksize * blocksize] = {0};
        // tmp = A*B
        for (int i = 0; i < blocksize; ++i) {
            for (int k = 0; k < blocksize; ++k) {
                for (int j = 0; j < blocksize; ++j) {
                    tmp[i * blocksize + j] += A[i * blocksize + k] * B[k * blocksize + j];
                }
            }
        }

        // dst = tmp*C
        for (int i = 0; i < blocksize; ++i) {
            for (int k = 0; k < blocksize; ++k) {
                for (int j = 0; j < blocksize; ++j) {
                    dst[i * blocksize + j] -= tmp[i * blocksize + k] * C[k * blocksize + j];
                }
            }
        }
    }

    template <class T, int blocksize>
    __global__ void
    cuInvertDiagonalAndFlatten(T* matNonZeroValues, int* rowIndices, int* colIndices, size_t numberOfRows, T* vec)
    {
        const auto row = blockDim.x * blockIdx.x + threadIdx.x;

        if (row < numberOfRows) {
            size_t nnzIdx = rowIndices[row];
            size_t nnzIdxLim = rowIndices[row + 1];

            // this loop will cause some extra checks that we are within the limit in the case of the diagonal having a
            // zero element
            while (colIndices[nnzIdx] != row && nnzIdx <= nnzIdxLim) {
                ++nnzIdx;
            }

            // diagBlock points to the start of where the diagonal block is stored
            T* diagBlock = &matNonZeroValues[blocksize * blocksize * nnzIdx];
            // vecBlock points to the start of the block element in the vector where the inverse of the diagonal block
            // element should be stored
            T* vecBlock = &vec[blocksize * blocksize * row];

            invBlockOutOfPlace<T, blocksize>(diagBlock, vecBlock);
        }
    }

    template <class T, int blocksize>
    __global__ void cuComputeLowerSolveLevelSet(T* mat,
                                                int* rowIndices,
                                                int* colIndices,
                                                int* indexConversion,
                                                int startIdx,
                                                int rowsInLevelSet,
                                                const T* dInv,
                                                const T* d,
                                                T* v)
    {
        const auto reorderedRowIdx = startIdx + (blockDim.x * blockIdx.x + threadIdx.x);
        if (reorderedRowIdx < rowsInLevelSet + startIdx) {

            const size_t nnzIdx = rowIndices[reorderedRowIdx];
            const int naturalRowIdx = indexConversion[reorderedRowIdx];

            T rhs[blocksize];
            for (int i = 0; i < blocksize; i++) {
                rhs[i] = d[naturalRowIdx * blocksize + i];
            }

            for (int block = nnzIdx; colIndices[block] < naturalRowIdx; ++block) {
                const int col = colIndices[block];
                mmv<T, blocksize>(&mat[block * blocksize * blocksize], &v[col * blocksize], rhs);
            }

            mv<T, blocksize>(&dInv[reorderedRowIdx * blocksize * blocksize], rhs, &v[naturalRowIdx * blocksize]);
        }
    }

    template <class T, int blocksize>
    __global__ void cuComputeUpperSolveLevelSet(T* mat,
                                                int* rowIndices,
                                                int* colIndices,
                                                int* indexConversion,
                                                int startIdx,
                                                int rowsInLevelSet,
                                                const T* dInv,
                                                T* v)
    {
        const auto reorderedRowIdx = startIdx + (blockDim.x * blockIdx.x + threadIdx.x);
        if (reorderedRowIdx < rowsInLevelSet + startIdx) {
            const size_t nnzIdxLim = rowIndices[reorderedRowIdx + 1];
            const int naturalRowIdx = indexConversion[reorderedRowIdx];

            T rhs[blocksize] = {0};

            for (int block = nnzIdxLim - 1; colIndices[block] > naturalRowIdx; --block) {
                const int col = colIndices[block];
                umv<T, blocksize>(&mat[block * blocksize * blocksize], &v[col * blocksize], rhs);
            }

            mmv<T, blocksize>(&dInv[reorderedRowIdx * blocksize * blocksize], rhs, &v[naturalRowIdx * blocksize]);
        }
    }

    template <class T, int blocksize>
    __global__ void cuComputeDiluDiagonal(T* mat,
                                          int* rowIndices,
                                          int* colIndices,
                                          int* reorderedToNatural,
                                          int* naturalToReordered,
                                          const int startIdx,
                                          int rowsInLevelSet,
                                          T* dInv)
    {
        const auto reorderedRowIdx = startIdx + blockDim.x * blockIdx.x + threadIdx.x;
        if (reorderedRowIdx < rowsInLevelSet + startIdx) {
            const int naturalRowIdx = reorderedToNatural[reorderedRowIdx];
            const size_t nnzIdx = rowIndices[reorderedRowIdx];

            int diagIdx = nnzIdx;
            while (colIndices[diagIdx] != naturalRowIdx) {
                ++diagIdx;
            }

            T dInvTmp[blocksize * blocksize];
            for (int i = 0; i < blocksize; ++i) {
                for (int j = 0; j < blocksize; ++j) {
                    dInvTmp[i * blocksize + j] = mat[diagIdx * blocksize * blocksize + i * blocksize + j];
                }
            }

            for (int block = nnzIdx; colIndices[block] < naturalRowIdx; ++block) {
                const int col = naturalToReordered[colIndices[block]];
                // find element with indices swapped
                // Binary search over block in the right row, [rowIndices[col], rowindices[col+1]-1] defines the range
                // we binary search over
                int left = rowIndices[col];
                int right = rowIndices[col + 1] - 1;
                int mid;

                while (left <= right) {
                    mid = left + (right - left) / 2; // overflow-safe average
                    const int col = colIndices[mid];

                    if (col == naturalRowIdx) {
                        break;
                    } else if (col < naturalRowIdx) {
                        left = mid + 1;
                    } else {
                        right = mid - 1;
                    }
                }

                const int symOpposite = mid;

                mmx2Subtraction<T, blocksize>(&mat[block * blocksize * blocksize],
                                              &dInv[col * blocksize * blocksize],
                                              &mat[symOpposite * blocksize * blocksize],
                                              dInvTmp);
            }

            invBlockInPlace<T, blocksize>(dInvTmp);

            for (int i = 0; i < blocksize; ++i) {
                for (int j = 0; j < blocksize; ++j) {
                    dInv[reorderedRowIdx * blocksize * blocksize + i * blocksize + j] = dInvTmp[i * blocksize + j];
                }
            }
        }
    }

    template <class T, int blocksize>
    __global__ void cuMoveDataToReordered(
        T* srcMatrix, int* srcRowIndices, T* dstMatrix, int* dstRowIndices, int* indexConversion, size_t numberOfRows)
    {
        const auto srcRow = blockDim.x * blockIdx.x + threadIdx.x;
        if (srcRow < numberOfRows) {

            const auto dstRow = indexConversion[srcRow];

            for (int srcBlock = srcRowIndices[srcRow], dstBlock = dstRowIndices[dstRow];
                 srcBlock < srcRowIndices[srcRow + 1];
                 ++srcBlock, ++dstBlock) {
                for (int i = 0; i < blocksize; ++i) {
                    for (int j = 0; j < blocksize; ++j) {
                        dstMatrix[dstBlock * blocksize * blocksize + i * blocksize + j]
                            = srcMatrix[srcBlock * blocksize * blocksize + i * blocksize + j];
                    }
                }
            }
        }
    }

    constexpr inline size_t getThreads([[maybe_unused]] size_t numberOfRows)
    {
        return 1024;
    }

    inline size_t getBlocks(size_t numberOfRows)
    {
        const auto threads = getThreads(numberOfRows);
        return (numberOfRows + threads - 1) / threads;
    }
} // namespace

template <class T, int blocksize>
void
invertDiagonalAndFlatten(T* mat, int* rowIndices, int* colIndices, size_t numberOfRows, T* vec)
{
    if (blocksize <= 3) {
        cuInvertDiagonalAndFlatten<T, blocksize>
            <<<getBlocks(numberOfRows), getThreads(numberOfRows)>>>(mat, rowIndices, colIndices, numberOfRows, vec);
    } else {
        OPM_THROW(std::invalid_argument, "Inverting diagonal is not implemented for blocksizes > 3");
    }
}

// perform the lower solve for all rows in the same level set
template <class T, int blocksize>
void
computeLowerSolveLevelSet(T* reorderedMat,
                          int* rowIndices,
                          int* colIndices,
                          int* indexConversion,
                          int startIdx,
                          int rowsInLevelSet,
                          const T* dInv,
                          const T* d,
                          T* v)
{
    cuComputeLowerSolveLevelSet<T, blocksize><<<getBlocks(rowsInLevelSet), getThreads(rowsInLevelSet)>>>(
        reorderedMat, rowIndices, colIndices, indexConversion, startIdx, rowsInLevelSet, dInv, d, v);
}

// perform the upper solve for all rows in the same level set
template <class T, int blocksize>
void
computeUpperSolveLevelSet(T* reorderedMat,
                          int* rowIndices,
                          int* colIndices,
                          int* indexConversion,
                          int startIdx,
                          int rowsInLevelSet,
                          const T* dInv,
                          T* v)
{
    cuComputeUpperSolveLevelSet<T, blocksize><<<getBlocks(rowsInLevelSet), getThreads(rowsInLevelSet)>>>(
        reorderedMat, rowIndices, colIndices, indexConversion, startIdx, rowsInLevelSet, dInv, v);
}

template <class T, int blocksize>
void
computeDiluDiagonal(T* reorderedMat,
                    int* rowIndices,
                    int* colIndices,
                    int* reorderedToNatural,
                    int* naturalToReordered,
                    const int startIdx,
                    int rowsInLevelSet,
                    T* dInv)
{
    if (blocksize <= 3) {
        cuComputeDiluDiagonal<T, blocksize>
            <<<getBlocks(rowsInLevelSet), getThreads(rowsInLevelSet)>>>(reorderedMat,
                                                                        rowIndices,
                                                                        colIndices,
                                                                        reorderedToNatural,
                                                                        naturalToReordered,
                                                                        startIdx,
                                                                        rowsInLevelSet,
                                                                        dInv);
    } else {
        OPM_THROW(std::invalid_argument, "Inverting diagonal is not implemented for blocksizes > 3");
    }
}

template <class T, int blocksize>
void
copyMatDataToReordered(
    T* srcMatrix, int* srcRowIndices, T* dstMatrix, int* dstRowIndices, int* naturalToReordered, size_t numberOfRows)
{
    cuMoveDataToReordered<T, blocksize><<<getBlocks(numberOfRows), getThreads(numberOfRows)>>>(
        srcMatrix, srcRowIndices, dstMatrix, dstRowIndices, naturalToReordered, numberOfRows);
}

#define INSTANTIATE_KERNEL_WRAPPERS(T, blocksize)                                                                      \
    template void invertDiagonalAndFlatten<T, blocksize>(T*, int*, int*, size_t, T*);                                  \
    template void copyMatDataToReordered<T, blocksize>(T*, int*, T*, int*, int*, size_t);                              \
    template void computeDiluDiagonal<T, blocksize>(T*, int*, int*, int*, int*, const int, int, T*);                   \
    template void computeUpperSolveLevelSet<T, blocksize>(T*, int*, int*, int*, int, int, const T*, T*);               \
    template void computeLowerSolveLevelSet<T, blocksize>(T*, int*, int*, int*, int, int, const T*, const T*, T*);

INSTANTIATE_KERNEL_WRAPPERS(float, 1);
INSTANTIATE_KERNEL_WRAPPERS(float, 2);
INSTANTIATE_KERNEL_WRAPPERS(float, 3);
INSTANTIATE_KERNEL_WRAPPERS(float, 4);
INSTANTIATE_KERNEL_WRAPPERS(float, 5);
INSTANTIATE_KERNEL_WRAPPERS(float, 6);
INSTANTIATE_KERNEL_WRAPPERS(double, 1);
INSTANTIATE_KERNEL_WRAPPERS(double, 2);
INSTANTIATE_KERNEL_WRAPPERS(double, 3);
INSTANTIATE_KERNEL_WRAPPERS(double, 4);
INSTANTIATE_KERNEL_WRAPPERS(double, 5);
INSTANTIATE_KERNEL_WRAPPERS(double, 6);
} // namespace Opm::cuistl::detail
